#include "hip/hip_runtime.h"

#include "kernel.h"
#include ""
#include "sm_60_atomic_functions.h"
#include <fstream>
#include <string>
#include <vector>
#include <mutex>
#include <thread>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>

using namespace std;
using namespace cudastatistics;

__global__ void filter_kernel(cudastatistics::StringArray input, size_t file_max_word_count, size_t word_max_len, char* output)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	int file_start = input.indices[idx];
	int file_end = input.indices[idx + 1];

	char* word = new char[word_max_len] { NULL };
	int word_len = 0;
	int word_count = 0;

	//NAGOT SKUMT

	for (int i = file_start; i < file_end; i++) {

		char ch = input.data[i];
		
		if (ch != ' ') {
			word[word_len] = ch;
			word_len++;			
		}
		else if (word_len > 0 ) {
			int output_idx = idx * file_max_word_count * word_max_len + word_count * word_max_len;
			memcpy(&output[output_idx], word, word_max_len * sizeof(char));
			word_count++;
			word_len = 0;
		}
	}

	if (word_len != 0) {
		int output_idx = idx * file_max_word_count * word_max_len + word_count * word_max_len;
		memcpy(&output[output_idx], word, word_max_len * sizeof(char));
	}

	delete[] word;
}

API bool cudastatistics::init(string& error) {
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		error = "hipSetDevice failed!Do you have a CUDA - capable GPU installed?";
		return false;
	}	
	return true;
}

API void build_content(const vector<string>& file_paths, string& contents, vector<int>& indices) {

	int thread_count = thread::hardware_concurrency();
	int files_per_thread = file_paths.size() / thread_count;
	contents = "a  sdqwe  d fpewro kfqwe123";
	indices = {0, 4, 13, (int)contents.size()};
	return;

	vector<thread> threads;
	mutex mutex;

	for (int i = 0; i < thread_count; i++) {

		int start = i * files_per_thread;
		int end = start + files_per_thread;

		thread thread([&] {

			for (int i = start; i < end; i++) {
				const string& path = file_paths[i];
				ifstream fstream(path);
				string content((istreambuf_iterator<char>(fstream)),
					istreambuf_iterator<char>());

				mutex.lock();
				indices.push_back(contents.size());
				contents += content;
				mutex.unlock();
			}

			});

		threads.push_back(move(thread));

	}

	for (thread& thread : threads) {
		thread.join();
	}
}

//https://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

API vector<string> cudastatistics::filter_files(const vector<string>& file_paths, int result_max_len, const SecretParams& params) 
{
	string files = "hej va fo  abc dfg     mammma apa ";
	vector<int> file_indices = { 0, 5, 10, (int)files.size() };

	StringArray input_arr{ nullptr, nullptr, file_indices.size() };

	hipMallocManaged(&input_arr.data, files.size() * sizeof(char));
	hipMallocManaged(&input_arr.indices, file_indices.size() * sizeof(int));

	memcpy(input_arr.data, &files[0], files.size() * sizeof(char));
	memcpy(input_arr.indices, &file_indices[0], file_indices.size() * sizeof(int));

	size_t thread_count = file_indices.size() - 1;
	size_t file_max_word_count = 5;
	size_t word_max_len = 10;

	char* result;
	hipMallocManaged(&result, thread_count * file_max_word_count * word_max_len * sizeof(char));

	filter_kernel << <1, thread_count >> > (input_arr, file_max_word_count, word_max_len, result);

	hipDeviceSynchronize();

	int size = thread_count * file_max_word_count * word_max_len;

	for (int i = 0; i < size; i ++) {

		cout << result[i];
	}

	return {};
}
